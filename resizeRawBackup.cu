#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include<opencv2/imgproc/types_c.h> 
// #include<opencv2/opencv.hpp>

cv::Mat getRgb(float *rawdata){
    int lines = 1020 * 3;
    int samples = 1020 * 3;
    int channels = 18;
    cv::Mat rgb_img,tmp_channel;
//    cv::Mat A(lines, samples, CV_32FC(18), rawdata);
//    std::vector<cv::Mat> spect_Channels(18);
//    cv::split(A, spect_Channels);
    std::vector<cv::Mat> rgb_channels;
//    float *tmp=rawdata;
//    rgb_channels.push_back(spect_Channels[10]);
//    rgb_channels.push_back(spect_Channels[7]);
//    rgb_channels.push_back(spect_Channels[1]);
//    time_t start_time = clock();
    for(int i =0 ;i<lines*samples*channels;i++){

        if(rawdata[i]<0)rawdata[i]=0;
        if(isnan(rawdata[i])){
            rawdata[i]=0;
        }
    }
//    time_t  end_time = clock();
//    std::cout << (end_time - start_time)/CLOCKS_PER_SEC  << std::endl;
    cv::Mat dst = cv::Mat(lines, samples, CV_8UC1);
    tmp_channel = cv::Mat(lines, samples, CV_32FC1,rawdata+1*lines*samples);
    cv::normalize(tmp_channel,dst,0,255,CV_MINMAX,CV_8UC1);
    rgb_channels.push_back(dst);

    dst = cv::Mat(lines, samples, CV_8UC1);
    tmp_channel = cv::Mat(lines, samples, CV_32FC1,rawdata+7*lines*samples);
    cv::normalize(tmp_channel,dst,0,255,CV_MINMAX,CV_8UC1);
    rgb_channels.push_back(dst);

    dst = cv::Mat(lines, samples, CV_8UC1);
    tmp_channel = cv::Mat(lines, samples, CV_32FC1,rawdata+10*lines*samples);
    cv::normalize(tmp_channel,dst,0,255,CV_MINMAX,CV_8UC1);
    rgb_channels.push_back(dst);

    cv::merge(rgb_channels, rgb_img);
    return rgb_img;
}


__global__ void trilinearInterpolation(float *input, float *output, int width, int height, int depth, int newWidth, int newHeight, int newDepth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < newWidth && y < newHeight && z < newDepth) {
        float xRatio = ((float)(width - 1)) / newWidth;
        float yRatio = ((float)(height - 1)) / newHeight;
        float zRatio = ((float)(depth - 1)) / newDepth;

        int x1 = (int)(x * xRatio);
        int y1 = (int)(y * yRatio);
        int z1 = (int)(z * zRatio);

        float xDiff = (x * xRatio) - x1;
        float yDiff = (y * yRatio) - y1;
        float zDiff = (z * zRatio) - z1;

        int index = z1 * width * height + y1 * width + x1;

        float c000 = input[index];
        float c001 = input[index + width * height];
        float c010 = input[index + width];
        float c011 = input[index + width * height + width];
        float c100 = input[index + 1];
        float c101 = input[index + width * height + 1];
        float c110 = input[index + width + 1];
        float c111 = input[index + width * height + width + 1];

        output[z * newWidth * newHeight + y * newWidth + x] =
            c000 * (1 - xDiff) * (1 - yDiff) * (1 - zDiff) +
            c100 * xDiff * (1 - yDiff) * (1 - zDiff) +
            c010 * (1 - xDiff) * yDiff* (1 - zDiff) +
            c001* (1-xDiff)*(1-yDiff)*zDiff+
            c101*xDiff*(1-yDiff)*zDiff+
            c011*(1-xDiff)*yDiff*zDiff+
            c110*xDiff*yDiff*(1-zDiff)+
            c111*xDiff*yDiff*zDiff;
    }
}

/*
template <int sharedSize>
__global__ void trilinearInterpolation(float *input, float *output, int width, int height, int depth, int newWidth, int newHeight, int newDepth) {
    // 定义共享内存数组
    __shared__ float sharedInput[sharedSize];
    __shared__ int beginindex;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int blockWidth = int((blockDim.x + 2 )/ 3);  // 向上取整 4
    int blockheight = int((blockDim.y + 2) / 3); // 4

    if (x < newWidth && y < newHeight && z < newDepth) {
        float xRatio = ((float)(width - 1)) / newWidth;
        float yRatio = ((float)(height - 1)) / newHeight;
        float zRatio = ((float)(depth - 1)) / newDepth;

        int x1 = (int)(x * xRatio);
        int y1 = (int)(y * yRatio);
        int z1 = (int)(z * zRatio);

        float xDiff = (x * xRatio) - x1;
        float yDiff = (y * yRatio) - y1;
        float zDiff = (z * zRatio) - z1;

        if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
            beginindex = z1 * width * height + y1 * width + x1;
        }
        __syncthreads();
        // int index = z1 * width * height + y1 * width + x1;

        // 加载数据到共享内存
        if (threadIdx.x < blockWidth && threadIdx.y < blockheight) {
            // 这里应该加载全局的
            int pos = beginindex + threadIdx.z * width * height +  threadIdx.y * width + threadIdx.x;
            sharedInput[threadIdx.z * blockWidth * blockheight +  threadIdx.y * blockWidth + threadIdx.x] = input[pos];  // 行 列 通道 排布
        }    
        // 同步线程
        __syncthreads();

        // 从共享内存中读取数据
         xRatio = ((float)(blockWidth - 1)) / blockDim.x;
         yRatio = ((float)(blockheight - 1)) / blockDim.y;
         zRatio = ((float)(blockDim.z - 1)) / blockDim.z;

         x1 = (int)(threadIdx.x * xRatio);
         y1 = (int)(threadIdx.y * yRatio);
         z1 = (int)(threadIdx.z * zRatio);

        int index = z1 * blockWidth * blockheight + y1 * blockWidth + x1;
        float c000 = sharedInput[index];
        float c001 = sharedInput[index + blockWidth * blockheight];
        float c010 = sharedInput[index + blockWidth];
        float c011 = sharedInput[index + blockWidth * blockheight + blockWidth];
        float c100 = sharedInput[index + 1];
        float c101 = sharedInput[index + blockWidth * blockheight + 1];
        float c110 = sharedInput[index + blockWidth + 1];
        float c111 = sharedInput[index + blockWidth * blockheight + blockWidth + 1];

        output[z * newWidth * newHeight + y * newWidth + x] =
            c000 * (1 - xDiff) * (1 - yDiff) * (1 - zDiff) +
            c100 * xDiff * (1 - yDiff) * (1 - zDiff) +
            c010 * (1 - xDiff) * yDiff* (1 - zDiff) +
            c001* (1-xDiff)*(1-yDiff)*zDiff+
            c101*xDiff*(1-yDiff)*zDiff+
            c011*(1-xDiff)*yDiff*zDiff+
            c110*xDiff*yDiff*(1-zDiff)+
            c111*xDiff*yDiff*zDiff;
    }
}
*/
void resize3D(float* input, float* output, int width, int height, int depth, int newWidth, int newHeight, int newDepth){
    float* d_input;
    float* d_output;
    hipMalloc(&d_input, width * height * depth * sizeof(float));
    hipMalloc(&d_output, newWidth * newHeight * newDepth * sizeof(float));
    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 记录 hipMemcpy 的开始时间
    hipEventRecord(start, 0);
    hipMemcpy(d_input, input, width * height * depth * sizeof(float), hipMemcpyHostToDevice);
    // 记录 hipMemcpy 的结束时间
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // 计算 hipMemcpy 的执行时间
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time to transfer data from host to device: " << elapsedTime << " ms" << std::endl;

    // dim3 blockSize(16, 16, 4);
    dim3 blockSize(12, 12, 6);
    dim3 gridSize((newWidth + blockSize.x - 1) / blockSize.x,
                  (newHeight + blockSize.y - 1) / blockSize.y,
                  (newDepth + blockSize.z - 1) / blockSize.z);
    hipEventRecord(start, 0); // int((blockSize.x + 2 / 3) * (blockSize.y + 2 / 3) * blockSize.z)
    const int sharedsize = 4 * 4 * 6;
    // 只能使用 extern 进行动态分配了！！
    trilinearInterpolation<<<gridSize, blockSize>>>(d_input, d_output, width, height, depth, newWidth, newHeight, newDepth);
    // 记录核函数的结束时间
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // 计算核函数的执行时间
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time to execute kernel: " << elapsedTime << " ms" << std::endl;

    hipEventRecord(start, 0);
    hipMemcpy(output, d_output, newWidth * newHeight * newDepth * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // 计算核函数的执行时间
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time to transfer data fro device to host:" << elapsedTime << " ms" << std::endl;
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    // 输入数据
    const int lines = 1020;     // 行数
    const int samples = 1020;   // 列数
    const int bands = 18;      // 波段数
    const int new_lines = lines * 3;     // 行数
    const int new_samples = samples * 3;   // 列数
    FILE* fp;
    fp = fopen("/public_data1/dataset_18ch/raw_data/20211021150620.raw", "rb");
    const int size = lines*samples*bands;
    float* data = new float[size];
    const int new_size = new_lines*new_samples*bands;
    float* output_data = new float[new_size];
    fread(data, sizeof(float), size, fp);
    // 下一步：流水线插值10张？
    resize3D(data, output_data, samples,lines, bands, new_samples, new_lines, bands);
    cv::Mat rgb = getRgb(output_data);
    cv::imwrite("resized_image_raw.jpg", rgb);
    delete[] data;
    delete[] output_data;
}