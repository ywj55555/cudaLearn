#include "hip/hip_runtime.h"
#include <iostream>
#include <dirent.h>
#include <string>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include<opencv2/imgproc/types_c.h> 
// 来一个宏定义 调用api看看是否成功，！！!
#define CHECK(call) \
do \
{ \
 const hipError_t error_code = call; \
 if (error_code != hipSuccess) \
 { \
 printf("CUDA Error:\\n"); \
 printf(" File: %s\\n", __FILE__); \
 printf(" Line: %d\\n", __LINE__); \
 printf(" Error code: %d\\n", error_code); \
 printf(" Error text: %s\\n", hipGetErrorString(error_code)); \
 exit(1); \
 } \
} while (0)

// #include<opencv2/opencv.hpp>
#define STREAM_NUM 4
cv::Mat getRgb(float *rawdata){
    int lines = 1020 * 3;
    int samples = 1020 * 3;
    int channels = 18;
    cv::Mat rgb_img,tmp_channel;
//    cv::Mat A(lines, samples, CV_32FC(18), rawdata);
//    std::vector<cv::Mat> spect_Channels(18);
//    cv::split(A, spect_Channels);
    std::vector<cv::Mat> rgb_channels;
//    float *tmp=rawdata;
//    rgb_channels.push_back(spect_Channels[10]);
//    rgb_channels.push_back(spect_Channels[7]);
//    rgb_channels.push_back(spect_Channels[1]);
//    time_t start_time = clock();
    for(int i =0 ;i<lines*samples*channels;i++){

        if(rawdata[i]<0)rawdata[i]=0;
        if(isnan(rawdata[i])){
            rawdata[i]=0;
        }
    }
//    time_t  end_time = clock();
//    std::cout << (end_time - start_time)/CLOCKS_PER_SEC  << std::endl;
    cv::Mat dst = cv::Mat(lines, samples, CV_8UC1);
    tmp_channel = cv::Mat(lines, samples, CV_32FC1,rawdata+1*lines*samples);
    cv::normalize(tmp_channel,dst,0,255,CV_MINMAX,CV_8UC1);
    rgb_channels.push_back(dst);

    dst = cv::Mat(lines, samples, CV_8UC1);
    tmp_channel = cv::Mat(lines, samples, CV_32FC1,rawdata+7*lines*samples);
    cv::normalize(tmp_channel,dst,0,255,CV_MINMAX,CV_8UC1);
    rgb_channels.push_back(dst);

    dst = cv::Mat(lines, samples, CV_8UC1);
    tmp_channel = cv::Mat(lines, samples, CV_32FC1,rawdata+10*lines*samples);
    cv::normalize(tmp_channel,dst,0,255,CV_MINMAX,CV_8UC1);
    rgb_channels.push_back(dst);

    cv::merge(rgb_channels, rgb_img);
    return rgb_img;
}


__global__ void trilinearInterpolation(float *input, float *output, int width, int height, int depth, int newWidth, int newHeight, int newDepth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < newWidth && y < newHeight && z < newDepth) {
        float xRatio = ((float)(width - 1)) / newWidth;
        float yRatio = ((float)(height - 1)) / newHeight;
        float zRatio = ((float)(depth - 1)) / newDepth;

        int x1 = (int)(x * xRatio);
        int y1 = (int)(y * yRatio);
        int z1 = (int)(z * zRatio);

        float xDiff = (x * xRatio) - x1;
        float yDiff = (y * yRatio) - y1;
        float zDiff = (z * zRatio) - z1;

        int index = z1 * width * height + y1 * width + x1;

        float c000 = input[index];
        float c001 = input[index + width * height];
        float c010 = input[index + width];
        float c011 = input[index + width * height + width];
        float c100 = input[index + 1];
        float c101 = input[index + width * height + 1];
        float c110 = input[index + width + 1];
        float c111 = input[index + width * height + width + 1];

        output[z * newWidth * newHeight + y * newWidth + x] =
            c000 * (1 - xDiff) * (1 - yDiff) * (1 - zDiff) +
            c100 * xDiff * (1 - yDiff) * (1 - zDiff) +
            c010 * (1 - xDiff) * yDiff* (1 - zDiff) +
            c001* (1-xDiff)*(1-yDiff)*zDiff+
            c101*xDiff*(1-yDiff)*zDiff+
            c011*(1-xDiff)*yDiff*zDiff+
            c110*xDiff*yDiff*(1-zDiff)+
            c111*xDiff*yDiff*zDiff;
    }
}

/*
template <int sharedSize>
__global__ void trilinearInterpolation(float *input, float *output, int width, int height, int depth, int newWidth, int newHeight, int newDepth) {
    // 定义共享内存数组
    __shared__ float sharedInput[sharedSize];
    __shared__ int beginindex;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int blockWidth = int((blockDim.x + 2 )/ 3);  // 向上取整 4
    int blockheight = int((blockDim.y + 2) / 3); // 4

    if (x < newWidth && y < newHeight && z < newDepth) {
        float xRatio = ((float)(width - 1)) / newWidth;
        float yRatio = ((float)(height - 1)) / newHeight;
        float zRatio = ((float)(depth - 1)) / newDepth;

        int x1 = (int)(x * xRatio);
        int y1 = (int)(y * yRatio);
        int z1 = (int)(z * zRatio);

        float xDiff = (x * xRatio) - x1;
        float yDiff = (y * yRatio) - y1;
        float zDiff = (z * zRatio) - z1;

        if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
            beginindex = z1 * width * height + y1 * width + x1;
        }
        __syncthreads();
        // int index = z1 * width * height + y1 * width + x1;

        // 加载数据到共享内存
        if (threadIdx.x < blockWidth && threadIdx.y < blockheight) {
            // 这里应该加载全局的
            int pos = beginindex + threadIdx.z * width * height +  threadIdx.y * width + threadIdx.x;
            sharedInput[threadIdx.z * blockWidth * blockheight +  threadIdx.y * blockWidth + threadIdx.x] = input[pos];  // 行 列 通道 排布
        }    
        // 同步线程
        __syncthreads();

        // 从共享内存中读取数据
         xRatio = ((float)(blockWidth - 1)) / blockDim.x;
         yRatio = ((float)(blockheight - 1)) / blockDim.y;
         zRatio = ((float)(blockDim.z - 1)) / blockDim.z;

         x1 = (int)(threadIdx.x * xRatio);
         y1 = (int)(threadIdx.y * yRatio);
         z1 = (int)(threadIdx.z * zRatio);

        int index = z1 * blockWidth * blockheight + y1 * blockWidth + x1;
        float c000 = sharedInput[index];
        float c001 = sharedInput[index + blockWidth * blockheight];
        float c010 = sharedInput[index + blockWidth];
        float c011 = sharedInput[index + blockWidth * blockheight + blockWidth];
        float c100 = sharedInput[index + 1];
        float c101 = sharedInput[index + blockWidth * blockheight + 1];
        float c110 = sharedInput[index + blockWidth + 1];
        float c111 = sharedInput[index + blockWidth * blockheight + blockWidth + 1];

        output[z * newWidth * newHeight + y * newWidth + x] =
            c000 * (1 - xDiff) * (1 - yDiff) * (1 - zDiff) +
            c100 * xDiff * (1 - yDiff) * (1 - zDiff) +
            c010 * (1 - xDiff) * yDiff* (1 - zDiff) +
            c001* (1-xDiff)*(1-yDiff)*zDiff+
            c101*xDiff*(1-yDiff)*zDiff+
            c011*(1-xDiff)*yDiff*zDiff+
            c110*xDiff*yDiff*(1-zDiff)+
            c111*xDiff*yDiff*zDiff;
    }
}
*/

void resize3D(std::string dirpath, int width, int height, int depth, int newWidth, int newHeight, int newDepth){
    // 每个流的执行顺序不保证把？？？就是执行引擎不保证执行完毕顺序和提交顺序一致？先实现再说！！
    float* d_input[STREAM_NUM];
    float* d_output[STREAM_NUM];
    float* h_input[STREAM_NUM];
    float* h_output[STREAM_NUM];
    int predictCount = 15;
    int inputSize = width * height * depth;
    int outputsize = newWidth * newHeight * newDepth;
    int nDevices;
    // 查询系统中可用的CUDA设备数量
    hipGetDeviceCount(&nDevices);
    hipStream_t stream[STREAM_NUM];
    for (int i = 0; i < STREAM_NUM; ++i) {
        CHECK(hipSetDevice(i % nDevices));
        CHECK(hipHostAlloc(&h_input[i], inputSize * sizeof(float), hipHostMallocDefault));  // hipHostMallocMapped 分配
        // h_input[i] = (float*)malloc(inputSize * sizeof(float));
        CHECK(hipMalloc(&d_input[i], inputSize * sizeof(float)));
        CHECK(hipHostAlloc(&h_output[i], outputsize * sizeof(float), hipHostMallocDefault));
        // h_output[i] = (float*)malloc(outputsize * sizeof(float));
        CHECK(hipMalloc(&d_output[i], outputsize * sizeof(float)));
        CHECK(hipStreamCreate(&stream[i]));
    }
    // for (int i = 0; i < STREAM_NUM; ++i) {
    //     hipHostAlloc(&h_input[i], inputSize * sizeof(float), hipHostMallocMapped);  // hipHostMallocMapped 分配
    //     // hipMalloc(&d_input[i], inputSize * sizeof(float));
    //     hipHostGetDevicePointer((void**)&d_input[i], (void*)h_input[i], 0);
    //     hipHostAlloc(&h_output[i], outputsize * sizeof(float), hipHostMallocMapped);
    //     // hipMalloc(&d_output[i], outputsize * sizeof(float));
    //     hipHostGetDevicePointer((void**)&d_output[i], (void*)h_output[i], 0);
    //     hipStreamCreate(&stream[i]);
    // }
    float elapsedTime;
    hipEvent_t start, stop;
    CHECK(hipSetDevice(0));
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    // 记录 hipMemcpy 的开始时间
    CHECK(hipEventRecord(start, 0));  // 绑定在默认流中！！这个和设备相关联把！！多设备如何确定？？

    DIR *dir;
    struct dirent *ent;
    std::string folder = dirpath;
    int count = 0;
    
    if ((dir = opendir(folder.c_str())) != NULL) {
        while ((ent = readdir(dir)) != NULL && count <  STREAM_NUM * 2) {
            std::string filename = ent->d_name;
            if (filename.size() > 4 && filename.substr(filename.size() - 4) == ".raw") {
                std::string filepath = folder + "/" + filename;
                FILE *file = fopen(filepath.c_str(), "rB");
                if (file == NULL) {
                    std::cout << "Error opening file: " << filepath << std::endl;
                } else {
                    // 得在不同device上进行实现
                    // 或者使用线程池
                    // 两种优化都试试
                    
                    std::cout << "Successfully opened file: " << filepath << std::endl;
                    int streamPos = count % STREAM_NUM;
                    if (count % STREAM_NUM == 0 && count != 0) {
                        for (int i = 0; i < STREAM_NUM; i++) {
                            CHECK(hipSetDevice(i % nDevices));
                            CHECK(hipStreamSynchronize(stream[i]));  // 需要访问输出结果，所以需要进行同步
                            cv::Mat rgb = getRgb(h_output[i]);
                            cv::imwrite("resized_image_raw" + std::to_string(count) + std::to_string(i) + ".jpg", rgb);
                        }
                    }
                    // fread之前需要确保之前的数据已经传输完毕了，但是需要用同步来保证把！！！
                    fread(h_input[streamPos], sizeof(float), inputSize, file);
                    CHECK(hipSetDevice(streamPos % nDevices));
                    CHECK(hipMemcpyAsync(d_input[streamPos], h_input[streamPos], width * height * depth * sizeof(float), hipMemcpyHostToDevice, stream[streamPos]));
                    dim3 blockSize(12, 12, 6);
                    dim3 gridSize((newWidth + blockSize.x - 1) / blockSize.x,
                                  (newHeight + blockSize.y - 1) / blockSize.y,
                                  (newDepth + blockSize.z - 1) / blockSize.z);
                    trilinearInterpolation<<<gridSize, blockSize, 0, stream[streamPos]>>>(d_input[streamPos], d_output[streamPos], width, height, depth, newWidth, newHeight, newDepth);
                    CHECK(hipMemcpyAsync(h_output[streamPos], d_output[streamPos], newWidth * newHeight * newDepth * sizeof(float), hipMemcpyDeviceToHost, stream[streamPos]));
                    
                    fclose(file);
                    count++;
                }
                
            }
        }
        closedir(dir);
        
    }
    // 最后同步所有的stream 注意在对应的设备上执行！！！
    for (int i = 0; i < STREAM_NUM; i++) {
       CHECK(hipSetDevice(i % nDevices));
        CHECK(hipStreamSynchronize(stream[i]));
        cv::Mat rgb = getRgb(h_output[i]);
        cv::imwrite("resized_image_raw" + std::to_string(count) + std::to_string(i) + ".jpg", rgb);
    }
    // hipMemcpy(d_input, input, width * height * depth * sizeof(float), hipMemcpyHostToDevice);
    // 记录 hipMemcpy 的结束时间
    CHECK(hipSetDevice(0));  
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    // 计算 hipMemcpy 的执行时间
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "all time: " << elapsedTime << " ms" << std::endl;
    for (int i = 0; i < STREAM_NUM; i++) {
        CHECK(hipSetDevice(i % nDevices));
       CHECK(hipHostFree(h_input[i]));
        // free(h_input[i]);
        CHECK(hipFree(d_input[i]));
        CHECK(hipHostFree(h_output[i]));
        // free(h_output[i]);
        CHECK(hipFree(d_output[i]));
        CHECK(hipStreamDestroy(stream[i]));
    }
    CHECK(hipSetDevice(0));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    // // dim3 blockSize(16, 16, 4);
    // dim3 blockSize(12, 12, 6);
    // dim3 gridSize((newWidth + blockSize.x - 1) / blockSize.x,
    //               (newHeight + blockSize.y - 1) / blockSize.y,
    //               (newDepth + blockSize.z - 1) / blockSize.z);
    // hipEventRecord(start, 0); // int((blockSize.x + 2 / 3) * (blockSize.y + 2 / 3) * blockSize.z)
    // const int sharedsize = 4 * 4 * 6;
    // // 只能使用 extern 进行动态分配了！！
    // trilinearInterpolation<sharedsize><<<gridSize, blockSize>>>(d_input, d_output, width, height, depth, newWidth, newHeight, newDepth);
    // // 记录核函数的结束时间
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // // 计算核函数的执行时间
    // hipEventElapsedTime(&elapsedTime, start, stop);
    // std::cout << "Time to execute kernel: " << elapsedTime << " ms" << std::endl;

    // hipEventRecord(start, 0);
    // hipMemcpy(output, d_output, newWidth * newHeight * newDepth * sizeof(float), hipMemcpyDeviceToHost);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // // 计算核函数的执行时间
    // hipEventElapsedTime(&elapsedTime, start, stop);
    // std::cout << "Time to transfer data fro device to host:" << elapsedTime << " ms" << std::endl;
    // hipFree(d_input);
    // hipFree(d_output);

}

int main() {
    // 输入数据
    const int lines = 1020;     // 行数
    const int samples = 1020;   // 列数
    const int bands = 18;      // 波段数
    const int new_lines = lines * 3;     // 行数
    const int new_samples = samples * 3;   // 列数
    // FILE* fp;
    std::string dirpath = "/public_data1/dataset_18ch/raw_data/";
    // fp = fopen("/public_data1/dataset_18ch/raw_data/20211021150620.raw", "rb");
    // const int size = lines*samples*bands;
    // float* data = new float[size];
    // const int new_size = new_lines*new_samples*bands;
    // float* output_data = new float[new_size];
    // fread(data, sizeof(float), size, fp);
    // 下一步：流水线插值10张？
    resize3D(dirpath, samples,lines, bands, new_samples, new_lines, bands);
    
    // delete[] data;
    // delete[] output_data;
}