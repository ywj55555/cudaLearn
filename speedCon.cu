#include <iostream>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

using namespace std;

__shared__ int sharedArray[64];

__global__ void add(int *a, const int *b){
    // const u32 tid =(blockIdx.x* blockDim.x)+ threadIdx.x;
    int i = blockIdx.x;
    a[i] += b[i];
    sharedArray[i] = 2 * a[i];
    if (i == 1)
    {
        printf("Value of shared_var: %d\n", sharedArray[i]);
    }
}

__global__ void add2(int *a, const int *b){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // 0 3 
    a[i] += b[i]; // 3 + 2 = 5
    // sharedArray[i] = 3 * a[i];
    // if (i == 1)
    // {
    //     printf("Value of shared_var: %d\n", sharedArray[i]);
    // }
}

int main(){
     const int N = 64; // number of elements
         int *a, *b, *temp, i;
       // malloc HOST memory for temp
       temp = new int [N];
           // malloc DEVICE memory for a, b
        hipMalloc(&a, N*sizeof(int));
        hipMalloc(&b, N*sizeof(int));
            // set a's values: a[i] = i
            for(i=0;i<N;i++) temp[i] = i;
        hipMemcpy(a, temp, N*sizeof(int), hipMemcpyHostToDevice);
             // set b's values: b[i] = 2*i
             for(i=0;i<N;i++) temp[i] = 2*i;
        hipMemcpy(b, temp, N*sizeof(int), hipMemcpyHostToDevice);
            // calculate a[i] += b[i] in GPU
        // cudaEvent_t start, stop;
        // cudaEventCreate(&start);
        // cudaEventCreate(&stop);
        // cudaEventRecord(start);
        // add<<<N,1>>>(a, b); // 已经修改a了！！
        // cudaEventRecord(stop);
        float elapsedTime;
        hipFuncAttributes attr;
        hipError_t err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(add));
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to get kernel attributes (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        printf("add Shared memory size: %zu bytes\n", attr.sharedSizeBytes);

        err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(add2));
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to get kernel attributes (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        printf("add2 Shared memory size: %zu bytes\n", attr.sharedSizeBytes);

        // cudaEventElapsedTime(&elapsedTime,start,stop);

        // printf("Time to generate: %.6f ms\n", elapsedTime);
        // cudaEventRecord(start);
        // add2<<<2,32>>>(a, b);
        // cudaEventRecord(stop);
        // cudaEventElapsedTime(&elapsedTime,start,stop);
        printf("Time to generate: %.6f ms\n", elapsedTime);
        // show a's values
        hipMemcpy(temp, a, N*sizeof(int), hipMemcpyDeviceToHost);
            //for(i=0;i<N;i++){
            // cout << temp[i] << endl;
            // }
        // free HOST & DEVICE memory
        delete [] temp;
        hipFree(a);
        hipFree(b);
}

