#include <iostream>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

using namespace std;

__global__ void add(int *a, const int *b){
            int i = blockIdx.x;
                a[i] += b[i];
}

__global__ void add2(int *a, const int *b){
            int i = blockIdx.x * blockDim.x + threadIdx.x;
                a[i] += b[i];
}

int main(){
     const int N = 64; // number of elements
         int *a, *b, *temp, i;
       // malloc HOST memory for temp
       temp = new int [N];
           // malloc DEVICE memory for a, b
        hipMalloc(&a, N*sizeof(int));
        hipMalloc(&b, N*sizeof(int));
            // set a's values: a[i] = i
            for(i=0;i<N;i++) temp[i] = i;
         hipMemcpy(a, temp, N*sizeof(int), hipMemcpyHostToDevice);
             // set b's values: b[i] = 2*i
             for(i=0;i<N;i++) temp[i] = 2*i;
        hipMemcpy(b, temp, N*sizeof(int), hipMemcpyHostToDevice);
            // calculate a[i] += b[i] in GPU
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
            add<<<N,1>>>(a, b);
        hipEventRecord(stop);
        float elapsedTime;

        hipEventElapsedTime(&elapsedTime,start,stop);

        printf("Time to generate: %.6f ms\n", elapsedTime);
        hipEventRecord(start);
            add2<<<2,32>>>(a, b);
        hipEventRecord(stop);
        hipEventElapsedTime(&elapsedTime,start,stop);
        printf("Time to generate: %.6f ms\n", elapsedTime);
        // show a's values
        hipMemcpy(temp, a, N*sizeof(int), hipMemcpyDeviceToHost);
            //for(i=0;i<N;i++){
            // cout << temp[i] << endl;
            // }
        // free HOST & DEVICE memory
        delete [] temp;
        hipFree(a);
        hipFree(b);
}

