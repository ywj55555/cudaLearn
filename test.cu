#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void add(int *a, const int *b)
{
	int i = blockIdx.x;
	a[i] += 2 * b[i];
}

int main()
{
	const int N = 10; // number of elements
	int *a, *b, *temp, i;
	// malloc HOST memory for temp
	temp = new int[N];
	// malloc DEVICE memory for a, b
	hipMalloc(&a, N * sizeof(int));
	hipMalloc(&b, N * sizeof(int));
	// set a's values: a[i] = i
	for (i = 0; i < N; i++)
		temp[i] = i;
	hipMemcpy(a, temp, N * sizeof(int), hipMemcpyHostToDevice);
	// set b's values: b[i] = 2*i
	for (i = 0; i < N; i++)
		temp[i] = 2 * i;
	hipMemcpy(b, temp, N * sizeof(int), hipMemcpyHostToDevice);
	// calculate a[i] += b[i] in GPU
	add<<<N, 1>>>(a, b);
	// show a's values
	hipMemcpy(temp, a, N * sizeof(int), hipMemcpyDeviceToHost);
	for (i = 0; i < N; i++)
	{
		cout << temp[i] << endl;
	}
	// free HOST & DEVICE memory
	delete[] temp;
	hipFree(a);
	hipFree(b);
}
